#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <string>
#include <vector>
#include <unordered_map>
#include <algorithm>
#include <chrono>

using namespace std;

const size_t CHUNK_SIZE = 1024 * 1024 * 16; // 16MB chunks
const int MAX_WORD_LENGTH = 50;
const int NUM_STREAMS = 48;

__device__ bool d_isspace(char c) {
  return c == ' ' || c == '\n' || c == '\r' || c == '\t';
}

__device__ char d_tolower(char c) {
  if (c >= 'A' && c <= 'Z') {
    return c + ('a' - 'A');
  }
  return c;
}

__device__ int d_strlen(const char *str) {
  int len = 0;
  while (str[len] != '\0') {
    len++;
  }
  return len;
}

__device__ bool d_strncmp(const char *s1, const char *s2, int n) {
  for (int i = 0; i < n; i++) {
    if (s1[i] != s2[i]) {
      return false;
    }
    if (s1[i] == '\0') {
      return true;
    }
  }
  return true;
}

__global__ void processWordsKernel(char *chunk, size_t chunkSize, char *words, int *wordLengths, int *wordCount, bool isFirstChunk) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= chunkSize) {
    return;
  }

  __shared__ char sharedChunk[1024];
  __shared__ bool isWordStart[1024];

  if (threadIdx.x < 1024 && idx < chunkSize) {
    sharedChunk[threadIdx.x] = chunk[idx];
    isWordStart[threadIdx.x] = (idx == 0) || d_isspace(chunk[idx - 1]);
  }
  __syncthreads();

  if (!isFirstChunk && idx == 0) {
    return;
  }

  if (isWordStart[threadIdx.x] && !d_isspace(sharedChunk[threadIdx.x])) {
    char word[MAX_WORD_LENGTH];
    int length = 0;

    while (idx + length < chunkSize && length < MAX_WORD_LENGTH - 1 && !d_isspace(chunk[idx + length])) {
      word[length] = d_tolower(chunk[idx + length]);
      length++;
    }

    if (idx + length >= chunkSize) {
      return;
    }

    word[length] = '\0';

    if (length > 3) {
      const char *suffixes[] = {"ing", "ed", "ly", "ful", "est", "ity", "es", "s"};
      for (const char *suffix : suffixes) {
        int suffixLen = d_strlen(suffix);
        if (length > suffixLen + 1 && d_strncmp(&word[length - suffixLen], suffix, suffixLen)) {
          length -= suffixLen;
          word[length] = '\0';
          break;
        }
      }
    }

    if (length > 0) {
      int wordIdx = atomicAdd(wordCount, 1);
      memcpy(&words[wordIdx * MAX_WORD_LENGTH], word, length + 1);
      wordLengths[wordIdx] = length;
    }
  }
}

int main() {
  auto start_time = chrono::high_resolution_clock::now();

  hipStream_t streams[NUM_STREAMS];
  for (int i = 0; i < NUM_STREAMS; i++) {
    hipStreamCreate(&streams[i]);
  }

  ifstream file("large_text.txt", ios::binary);
  if (!file) {
    cerr << "Error opening file\n";
    return 1;
  }

  file.seekg(0, ios::end);
  size_t fileSize = file.tellg();
  file.seekg(0, ios::beg);

  char *h_chunk, *d_chunk;
  char *d_words;
  int *d_wordLengths, *d_wordCount;

  hipHostMalloc(&h_chunk, CHUNK_SIZE, hipHostMallocDefault);
  hipMalloc(&d_chunk, CHUNK_SIZE);
  hipMalloc(&d_words, CHUNK_SIZE * MAX_WORD_LENGTH);
  hipMalloc(&d_wordLengths, CHUNK_SIZE * sizeof(int));
  hipMalloc(&d_wordCount, sizeof(int));

  unordered_map<string, int> wordFrequency;
  int currentStream = 0;

  float process_time = 0;

  for (size_t offset = 0; offset < fileSize; offset += CHUNK_SIZE) {
    auto chunk_start = chrono::high_resolution_clock::now();

    size_t currentChunkSize = min(CHUNK_SIZE, fileSize - offset);

    file.read(h_chunk, currentChunkSize);

    hipMemcpyAsync(d_chunk, h_chunk, currentChunkSize, hipMemcpyHostToDevice, streams[currentStream]);

    hipMemsetAsync(d_wordCount, 0, sizeof(int), streams[currentStream]);

    int blockSize = 256;
    int numBlocks = (currentChunkSize + blockSize - 1) / blockSize;

    processWordsKernel<<<numBlocks, blockSize, 0, streams[currentStream]>>>(d_chunk, currentChunkSize, d_words, d_wordLengths, d_wordCount, offset == 0);

    int h_wordCount;
    hipMemcpyAsync(&h_wordCount, d_wordCount, sizeof(int), hipMemcpyDeviceToHost, streams[currentStream]);

    hipStreamSynchronize(streams[currentStream]);

    char *h_words = new char[h_wordCount * MAX_WORD_LENGTH];
    int *h_wordLengths = new int[h_wordCount];

    hipMemcpy(h_words, d_words, h_wordCount * MAX_WORD_LENGTH, hipMemcpyDeviceToHost);
    hipMemcpy(h_wordLengths, d_wordLengths, h_wordCount * sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < h_wordCount; i++) {
      string word(&h_words[i * MAX_WORD_LENGTH]);
      wordFrequency[word]++;
    }

    delete[] h_words;
    delete[] h_wordLengths;

    currentStream = (currentStream + 1) % NUM_STREAMS;

    auto chunk_end = chrono::high_resolution_clock::now();
    process_time += chrono::duration<float>(chunk_end - chunk_start).count();

    float progress = (float)offset / fileSize * 100;
    cout << "\rProgress: " << progress << "%" << flush;
  }

  vector<pair<string, int>> wordList;
  for (const auto &pair : wordFrequency) {
    wordList.push_back(pair);
  }

  sort(wordList.begin(), wordList.end(), [](const auto &a, const auto &b) {
    return a.second > b.second;
  });

  auto end_time = chrono::high_resolution_clock::now();
  float total_time = chrono::duration<float>(end_time - start_time).count();

  cout << "\n\nPerformance Statistics:\n";
  cout << "----------------------\n";
  cout << "Total time: " << total_time << " seconds\n";

  cout << "\n\nWord frequencies:\n";
  cout << "Word\t\tCount\n";
  cout << "-------------------\n";
  for (const auto &pair : wordList) {
    cout << pair.first << "\t\t" << pair.second << "\n";
  }

  hipHostFree(h_chunk);
  hipFree(d_chunk);
  hipFree(d_words);
  hipFree(d_wordLengths);
  hipFree(d_wordCount);

  for (int i = 0; i < NUM_STREAMS; i++) {
    hipStreamDestroy(streams[i]);
  }

  return 0;
}

// Compile: "C:\Program Files\Microsoft Visual Studio\2022\Community\VC\Auxiliary\Build\vcvars64.bat"
//          nvcc -O3 -arch=sm_89 -std=c++17 WordCountCU.cu -o WordCountCU.exe
// Run:     WordCountCU.exe